#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <filesystem>

namespace fs = std::filesystem;

#define THREADS_PER_BLOCK 256

// CUDA kernel simple: invertir colores
__global__ void invertColors(unsigned char* d_img, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_img[idx] = 255 - d_img[idx];
    }
}

void processImage(const std::string& input_path, const std::string& output_path) {
cv::Mat img = cv::imread(input_path, cv::IMREAD_COLOR);
    if (img.empty()) {
        printf("No se pudo leer la imagen %s\n", input_path.c_str());
        return;
    }

    int size = img.rows * img.cols;
    unsigned char* d_img;

    hipMalloc(&d_img, size);
    hipMemcpy(d_img, img.data, size, hipMemcpyHostToDevice);

    int blocks = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    invertColors<<<blocks, THREADS_PER_BLOCK>>>(d_img, size);
    hipMemcpy(img.data, d_img, size, hipMemcpyDeviceToHost);

    hipFree(d_img);

    cv::imwrite(output_path, img);
    printf("Procesada: %s -> %s\n", input_path.c_str(), output_path.c_str());
}

int main() {
    std::string input_dir = "data/";
    std::string output_dir = "output/";

    for (const auto& entry : fs::directory_iterator(input_dir)) {
        if (entry.is_regular_file()) {
            std::string filename = entry.path().filename().string();
            std::string input_path = input_dir + filename;
            
            // Convertir salida a PNG
            std::string output_name = entry.path().stem().string() + ".png";
            std::string output_path = output_dir + output_name;
            
            processImage(input_path, output_path);
        }
    }

    return 0;
}

